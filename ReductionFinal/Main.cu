#include "hip/hip_runtime.h"
#include "functions.h"

const int BLOCK_SIZE = 256;

__global__ void sum(float* input, float* output, int size)
{
	__shared__ float partialSum[2 * BLOCK_SIZE];
	unsigned int t = threadIdx.x, start = 2 * blockIdx.x * blockDim.x;
	
	if (start + t < size)
		partialSum[t] = input[start + t];
	else
		partialSum[t] = 0;
	if (start + blockDim.x + t < size)
		partialSum[blockDim.x + t] = input[start + blockDim.x + t];
	else
		partialSum[blockDim.x + t] = 0;

	for (unsigned int stride = 1; stride <= blockDim.x; stride <<= 1) {
		__syncthreads();
		if (t % stride == 0)
			partialSum[2*t] += partialSum[2*t + stride];
	}
	__syncthreads();
	if (t == 0)
		output[blockIdx.x] += partialSum[0];

	

}
void reductionOnGpu(float* h_A, float* h_R, int size) {
	
	sum <<<1, BLOCK_SIZE >>> (h_A, h_R, size);

	//hipMemcpy(R, h_A, sizeof(float), hipMemcpyDeviceToHost);
	
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
	int main(int argc, char** argv) {
		rand_init();
		int size;
		printf("Enter length of vector: ");
		scanf("%d", &size);

		float* A = new float[size];
		fill(A, size);
		printf("***Reduction on CPU***");
		StopWatchInterface* timer = 0;
		sdkCreateTimer(&timer);
		sdkStartTimer(&timer);

		float result = sumArray(A, size);

		sdkStopTimer(&timer);
		printf("Reduction time on CPU: %f (ms)\n", sdkGetTimerValue(&timer));
		sdkDeleteTimer(&timer);

		float* D_A = (float*)malloc(sizeof(float) * size);
		for (int i = 0; i < size; i++) {
			D_A[i] = A[i];
		}

		printf("***Reduction on GPU***\n");
		sdkCreateTimer(&timer);
		sdkStartTimer(&timer);
		
		float D_result = divide(D_A, size);

		sdkStopTimer(&timer);
		printf("Reduction time on GPU: %f (ms)\n", sdkGetTimerValue(&timer));
		sdkDeleteTimer(&timer);

		if (compare(D_result, result)) {
			printf("Results match");
		}
		else {
			printf("Results does not match. CPU: %f GPU: %f", result, D_result);
		}
		
		getchar();
		getchar();
		hipFree(D_A);
		delete[] A;
		return 0;
	}