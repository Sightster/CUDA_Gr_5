﻿// Testowy.cpp : Defines the entry point for the application.
//

#include "Main.h"


int main(int argc, int argv)
{
    bool cpu, gpu, timeMeasure;
    cpu = true;
    gpu = true;
    timeMeasure = true;
    int vectors, matrices;
    vectors = matrices = 5;

    std::cout << "Current program configuration:\n";
    std::cout << "CPU: " << (cpu ? "ON" : "OFF") << " | GPU: " << (gpu ? "ON" : "OFF")
        << " | Time measure: " << (timeMeasure ? "ON\n" : "OFF\n");
    std::cout << "Workspaces size Vectors: " << vectors << " Matrices: " << matrices << '\n';

    bool conf;
    std::cout << "Do you want change size of Workspaces? 0)No     1)Yes\n";
    std::cin >> conf;
    while (std::cin.fail()) {
        std::cout << "Enter 0 or 1 \n";
        std::cin.clear();
        std::cin.ignore(100, '\n');
        std::cin >> conf;
    }
    if (conf) {
        std::cout << "Enter number of vectors: ";
        std::cin >> vectors;
        std::cout << "Enter number of matrices: ";
        std::cin >> matrices;
    }
    
    Manager core(cpu, gpu, timeMeasure, vectors, matrices);
    core.mainMenu();
    
    return 0;
}


