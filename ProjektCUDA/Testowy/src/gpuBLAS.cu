#include "hip/hip_runtime.h"
#include "gpuBLAS.h"


__global__ void addKernel(int* A, int* B, int* R, int size) {
    int stride = blockDim.x * gridDim.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    for (; i < size; i+=stride) {
        R[i] = A[i] + B[i];
    }
}

__global__ void subKernel(int* A, int* B, int* R, int size) {
    int stride = blockDim.x * gridDim.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    for (; i < size; i += stride) {
        R[i] = A[i] - B[i];
    }
}

__global__ void dotKernel(int* A, int* B, int* R, int size) {
    int stride = blockDim.x * gridDim.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    for (; i < size; i += stride) {
        R[i] = A[i] * B[i];
    }
}

__global__ void mulKernel(int* A, int* B, int* R, int m, int n, int k) {
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    int sum = 0;
    if ((Row < m) && (Col < k)) {
        
        for (int i = 0; i < n; i++) {
            sum += A[Row * n + i] * B[i * k + Col];
        }
        R[Row * k + Col] = sum;
    }
}

void addGpu(Gpu* GPU,int* h_A, int* h_B, int* h_R, int n) {
    int* d_A, * d_B, * d_R;
    int size = n * sizeof(int);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_R, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    addKernel << <GPU->getGrid(), GPU->getBlock() >> > (d_A, d_B, d_R, n);
    hipDeviceSynchronize();
    hipMemcpy(h_R, d_R, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_R);
 
}

void subGpu(Gpu* GPU, int* h_A, int* h_B, int* h_R, int n) {
    int* d_A, * d_B, * d_R;
    int size = n * sizeof(int);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_R, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    subKernel << <GPU->getGrid(), GPU->getBlock() >> > (d_A, d_B, d_R, n);
    hipDeviceSynchronize();
    hipMemcpy(h_R, d_R, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_R);
}

int dotGpu(Gpu* GPU, int* h_A, int* h_B, int n) {
    int* d_A, * d_B, * d_R;
    int* h_R = new int[n];
    int size = n * sizeof(int);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_R, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dotKernel << <GPU->getGrid(), GPU->getBlock() >> > (d_A, d_B, d_R, n);
    hipDeviceSynchronize();
    hipMemcpy(h_R, d_R, size, hipMemcpyDeviceToHost);
    
    int sum = 0;
    for (int i = 0; i < n; i++) {
        sum += h_R[i];
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_R);
    delete h_R;

    return sum;
}

void mulGpu(Gpu* GPU, int* h_A, int* h_B, int* h_R, int m,int n, int k) {
    int* d_A, * d_B, * d_R;
    const int BLOCK_SIZE = 16;
    hipMalloc((void**)&d_A, m * n * sizeof(int));
    hipMalloc((void**)&d_B, n * k * sizeof(int));
    hipMalloc((void**)&d_R, m * k * sizeof(int));

    hipMemcpy(d_A, h_A, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * k * sizeof(int), hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    mulKernel << <dimGrid, dimBlock >> > (d_A, d_B, d_R, m, n, k);
    hipDeviceSynchronize();
    hipMemcpy(h_R, d_R, m * k * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_R);

}

StopWatchInterface*  startTimer() {
    StopWatchInterface* timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
    return timer;
}

void stopTimer(StopWatchInterface* timer) {
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
}





