#include "Gpu.h"

void Gpu::changeSizes() {
    do {
        std::cout << "Enter grid size:\n";
        std::cin.clear();
        std::cin.ignore(100, '\n');
        std::cin >> _gridSize;
    } while (std::cin.fail());

    do {
        std::cout << "Enter block size:\n";
        std::cin.clear();
        std::cin.ignore(100, '\n');
        std::cin >> _blockSize;
    } while (std::cin.fail());
}

int Gpu::getGrid() {
    return _gridSize;
}

int Gpu::getBlock() {
    return _blockSize;
}

Gpu::Gpu() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
        prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
        prop.memoryBusWidth);
    _gridSize = prop.multiProcessorCount;
    _blockSize = prop.maxThreadsPerBlock;
    printf("  Multi Processor Count : %d\n",
        _gridSize);
    printf("  Max threads per block : %d\n\n",
        _blockSize);
    
}