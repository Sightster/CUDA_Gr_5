#include "hip/hip_runtime.h"
#include"MWorkspace.h"

void mWorkspace::intInput(int* input)const {
	std::cin >> *input;
	while (std::cin.fail()) {
		std::cout << "Enter integer number: \n";
		std::cin.clear();
		std::cin.ignore(100, '\n');
		std::cin >> *input;
	}
}

bool mWorkspace::inputPosition(int* position1) const {
	intInput(position1);
	if (*position1 >= _size || *position1 < 0) {
		std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
		return false;
	}
	if (_ptr[*position1] == nullptr) {
		std::cout << "This position doesn't contain vector\n";
		return false;
	}
	return true;
}

Matrix* mWorkspace::createMatrix() {
	int position, row, col;
	std::cout << "In which position: ";
	intInput(&position);
	std::cout << "How many rows new matrix has: ";
	intInput(&row);
	std::cout << "How many columns new matrix has: ";
	intInput(&col);

	if (position >= _size || position < 0) {
		std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
		return nullptr;
	}

	if (_ptr[position] != nullptr) {
		bool replace;
		std::cout << "Given position is already taken!\n" << "Do you want replace matrix nr " << position << "?\n"
			<< "Press: 0)No     1)Yes";
		std::cin >> replace;
		if (replace == 0)
			return _ptr[position];
	}
	delete _ptr[position];

	_ptr[position] = new Matrix(row, col);
	if (_ptr[position] == nullptr)
		std::cout << "Memory alocation failed\n";
	else
		std::cout << "New matrix created successfully\n";
	_ptr[position]->fill(1, 5);
	return _ptr[position];
}

void mWorkspace::deleteMatrix() {
	int position;
	std::cout << "Which matrix do you want to delete: ";
	intInput(&position);
	if (position >= _size || position < 0) {
		std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
		return;
	}
	delete _ptr[position];
	_ptr[position] = nullptr;
	std::cout << "Matrix deleted successfully.\n";
}

void mWorkspace::fillRandom() {
	int min, max, position;
	std::cout << "Which matrix do you want to fill: ";
	intInput(&position);
	if (position >= _size || position < 0) {
		std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
		return;
	}
	if (_ptr[position] == nullptr) {
		std::cout << "This position doesn't contain matrix\n";
		return;
	}

	std::cout << "Provide bottom limit: ";
	intInput(&min);
	std::cout << "Provide upper limit: ";
	intInput(&max);
	if (min > max) {
		std::cout << "Upper limit should be greater than bottom one!\n";
		return;
	}

	_ptr[position]->fill(min, max);
}

void mWorkspace::show() const {
	std::cout << "Actual state of mWorkspace:\n";
	for (int i = 0; i < _size; i++) {
		std::cout << "Postion " << i << ": ";
		if (_ptr[i] == nullptr) {
			std::cout << "Empty\n";
		}
		else {
			_ptr[i]->show();
		}
	}
}

void mWorkspace::printMatrix() const {
	int position;
	std::cout << "In which position: ";
	intInput(&position);
	if (position >= _size || position < 0) {
		std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
		return;
	}

	std::cout << "Matrix nr " << position << ": \n";
	if (_ptr[position] == nullptr) {
		std::cout << "Empty\n";
		return;
	}
	_ptr[position]->print();
}

void mWorkspace::compareMatrices() const {
	int position1, position2;
	std::cout << "Which matrix do you want to compare: ";
	if (!inputPosition(&position1))
		return;

	std::cout << "Witch which matrix: ";
	if (!inputPosition(&position2))
		return;

	if ((*_ptr[position1]) == (*_ptr[position2]))
		std::cout << "Matrix " << position1 << " and matrix " << position2 << " are equal.\n";
	else
		std::cout << "Matrix " << position1 << " and matrix " << position2 << " are different.\n";
}

void mWorkspace::addMatrices(bool cpu, bool gpu, bool timeMeasure) const {
	int position1, position2, position3, position4;
	std::cout << "Which matrix do you want to add: ";
	if (!inputPosition(&position1))
		return;

	std::cout << "With which matrix: ";
	if (!inputPosition(&position2))
		return;
	if (cpu) {
		std::cout << "Chose position in workspace where result will be save: ";
		intInput(&position3);

		if (position3 >= _size || position3 < 0) {
			std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
			return;
		}
		if (_ptr[position3] != nullptr) {
			std::cout << "Given position is already taken!\n";
			return;
		}
	}

	if (gpu) {
		std::cout << "Chose position in workspace where result from GPU will be save: ";
		intInput(&position4);

		if (position4 >= _size || position4 < 0) {
			std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
			return;
		}
		if (_ptr[position4] != nullptr) {
			std::cout << "Given position is already taken!\n";
			return;
		}
	}

	if (cpu) {
		StopWatchInterface* timer;
		if (timeMeasure) {
			printf("Cpu computing...\n");
			timer = startTimer();
		}
		_ptr[position3] = new Matrix(_ptr[position2]->getRow(), _ptr[position1]->getCol());
		*_ptr[position3] = *_ptr[position1] + *_ptr[position2];
		if (timeMeasure) {
			printf("Computing ended\n");
			stopTimer(timer);
			putchar('\n');
		}
	}

	if (gpu) {
		StopWatchInterface* timer;
		if (timeMeasure) {
			printf("Gpu computing...\n");
			timer = startTimer();
		}
		_ptr[position4] = new Matrix(_ptr[position2]->getRow(), _ptr[position1]->getCol());
		addGpu(_GPU, _ptr[position1]->getPtr(), _ptr[position2]->getPtr(), _ptr[position4]->getPtr(),
			_ptr[position1]->getRow() * _ptr[position1]->getCol());
		if (timeMeasure) {
			printf("Computing ended\n");
			stopTimer(timer);
			putchar('\n');
		}
	}

	if (gpu && cpu) {
		if (*_ptr[position4] == *_ptr[position3])
			printf("Results match!\n");
		else
			printf("Results does not match!\n");
	}
}

void mWorkspace::substractMatrices(bool cpu, bool gpu, bool timeMeasure) const {
	int position1, position2, position3, position4;
	std::cout << "Which matrix do you want to substract: ";
	if (!inputPosition(&position1))
		return;

	std::cout << "With which matrix: ";
	if (!inputPosition(&position2))
		return;
	if (cpu) {
		std::cout << "Chose position in workspace where result will be save: ";
		intInput(&position3);

		if (position3 >= _size || position3 < 0) {
			std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
			return;
		}
		if (_ptr[position3] != nullptr) {
			std::cout << "Given position is already taken!\n";
			return;
		}
	}

	if (gpu) {
		std::cout << "Chose position in workspace where result from GPU will be save: ";
		intInput(&position4);

		if (position4 >= _size || position4 < 0) {
			std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
			return;
		}
		if (_ptr[position4] != nullptr) {
			std::cout << "Given position is already taken!\n";
			return;
		}
	}

	if (cpu) {
		StopWatchInterface* timer;
		if (timeMeasure) {
			printf("Cpu computing...\n");
			timer = startTimer();
		}
		_ptr[position3] = new Matrix(_ptr[position2]->getRow(), _ptr[position1]->getCol());
		*_ptr[position3] = *_ptr[position1] - *_ptr[position2];
		if (timeMeasure) {
			printf("Computing ended\n");
			stopTimer(timer);
			putchar('\n');
		}
	}

	if (gpu) {
		StopWatchInterface* timer;
		if (timeMeasure) {
			printf("Gpu computing...\n");
			timer = startTimer();
		}
		_ptr[position4] = new Matrix(_ptr[position2]->getRow(), _ptr[position1]->getCol());
		subGpu(_GPU, _ptr[position1]->getPtr(), _ptr[position2]->getPtr(), _ptr[position4]->getPtr(),
			_ptr[position1]->getRow() * _ptr[position1]->getCol());
		if (timeMeasure) {
			printf("Computing ended\n");
			stopTimer(timer);
			putchar('\n');
		}
	}

	if (gpu && cpu) {
		if (*_ptr[position4] == *_ptr[position3])
			printf("Results match!\n");
		else
			printf("Results does not match!\n");
	}
}


void mWorkspace::multiplyMatrices(bool cpu, bool gpu, bool timeMeasure) const {
	int position1, position2, position3, position4;
	std::cout << "Which matrix do you want to multiply: ";
	if (!inputPosition(&position1))
		return;

	std::cout << "With which matrix: ";
	if (!inputPosition(&position2))
		return;
	if (cpu) {
		std::cout << "Chose position in workspace where result will be save: ";
		intInput(&position3);

		if (position3 >= _size || position3 < 0) {
			std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
			return;
		}
		if (_ptr[position3] != nullptr) {
			std::cout << "Given position is already taken!\n";
			return;
		}
	}

	if (gpu) {
		std::cout << "Chose position in workspace where result from GPU will be save: ";
		intInput(&position4);

		if (position4 >= _size || position4 < 0) {
			std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
			return;
		}
		if (_ptr[position4] != nullptr) {
			std::cout << "Given position is already taken!\n";
			return;
		}
	}

	if (cpu) {
		StopWatchInterface* timer;
		if (timeMeasure) {
			printf("Cpu computing...\n");
			timer = startTimer();
		}
		_ptr[position3] = new Matrix(_ptr[position1]->getCol(), _ptr[position2]->getRow());
		*_ptr[position3] = *_ptr[position1] * *_ptr[position2];
		if (timeMeasure) {
			printf("Computing ended\n");
			stopTimer(timer);
			putchar('\n');
		}
	}

	if (gpu) {
		StopWatchInterface* timer;
		if (timeMeasure) {
			printf("Cpu computing...\n");
			timer = startTimer();
		}
		_ptr[position4] = new Matrix(_ptr[position1]->getRow(), _ptr[position2]->getCol());
		mulGpu(_GPU, _ptr[position1]->getPtr(), _ptr[position2]->getPtr(), _ptr[position4]->getPtr(),
			_ptr[position1]->getRow(), _ptr[position1]->getCol(), _ptr[position2]->getCol());
		if (timeMeasure) {
			printf("Computing ended\n");
			stopTimer(timer);
			putchar('\n');
		}
	}

	if (gpu && cpu) {
		if (*_ptr[position4] == *_ptr[position3])
			printf("Results match!\n");
		else
			printf("Results does not match!\n");
	}
}

void mWorkspace::invertMatrices(bool cpu, bool gpu, bool timeMeasure) const {

}

mWorkspace::mWorkspace(int size, Gpu* gpu) :
	_size(size),
	_ptr(new Matrix* [size]),
	_GPU(gpu)
{
	for (int i = 0; i < size; i++) {
		_ptr[i] = nullptr;
	}
}

mWorkspace::~mWorkspace() {
	for (int i = 0; i < _size; i++) {
		delete _ptr[i];
	}
	delete[] _ptr;
}