#include "hip/hip_runtime.h"
#include "VWorkspace.h"
//vWorkspace

void vWorkspace::intInput(int* input)const {
	std::cin >> *input;
	while (std::cin.fail()) {
		std::cout << "Enter integer number: \n";
		std::cin.clear();
		std::cin.ignore(100, '\n');
		std::cin >> *input;
	}
}

bool vWorkspace::inputPosition(int* position1) const {
	intInput(position1);
	if (*position1 >= _size || *position1 < 0) {
		std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
		return false;
	}
	if (_ptr[*position1] == nullptr) {
		std::cout << "This position doesn't contain vector\n";
		return false;
	}
	return true;
}

Vector* vWorkspace::createVector() {
	int position, size;
	std::cout << "In which position: ";
	intInput(&position);
	
	if (position >= _size || position < 0) {
		std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
		return nullptr;
	}
	if (_ptr[position] != nullptr) {
		bool replace;
		std::cout << "Given position is already taken!\n" << "Do you want replace vector nr " << position << "?\n"
			<< "Press: 0)No     1)Yes\n";
		std::cin >> replace;
		if(replace == 0)
			return _ptr[position];
	}

	std::cout << "Enter size of new vector: ";
	intInput(&size);
	delete _ptr[position];
	_ptr[position] = new Vector(size);
	if (_ptr[position] == nullptr)
		std::cout << "Memory alocation failed\n";
	else
		std::cout << "New vector created successfully\n";
	return _ptr[position];
}

void vWorkspace::deleteVector() {
	int position;
	std::cout << "Which vector do you want to delete: ";
	intInput(&position);
	if (position >= _size || position < 0) {
		std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
		return;
	}
	delete _ptr[position];
	_ptr[position] = nullptr;
	std::cout << "Vector deleted successfully.\n";
}

void vWorkspace::fillRandom(){
	int min, max, position;
	std::cout << "Which vector do you want to fill: ";
	if (!inputPosition(&position))
		return;

	std::cout << "Provide bottom limit: ";
	intInput(&min);
	std::cout << "Provide upper limit: ";
	intInput(&max);
	if (min > max) {
		std::cout << "Upper limit should be greater than bottom one!\n";
		return;
	}

	_ptr[position]->fill(min, max);
}

void vWorkspace::show() const{
	std::cout << "Actual state of vWorkspace:\n";
	for (int i = 0; i < _size; i++) {
		std::cout << "Position " << i << ": ";
		if (_ptr[i] == nullptr) {
			std::cout << "Empty\n";
		}
		else {
			_ptr[i]->show();
		}
	}
}

void vWorkspace::printVector() const {
	int position;
	std::cout << "Which vector: ";
	std::cin >> position;
	if (position >= _size || position < 0) {
		std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
		return;
	}

	std::cout << "Vector nr " << position << ": ";
	if (_ptr[position] == nullptr) {
		std::cout << "Empty\n";
		return;
	}
	_ptr[position]->print();
}

void vWorkspace::compareVectors() const {
	int position1, position2;
	std::cout << "Which vector do you want to compare: ";
	if (!inputPosition(&position1))
		return;

	std::cout << "With which vector: ";
	if (!inputPosition(&position2))
		return;

	if ((*_ptr[position1]) == (*_ptr[position2]))
		std::cout << "Vector " << position1 << " and vector " << position2 << " are equal.\n";
	else
		std::cout << "Vector " << position1 << " and vector " << position2 << " are different.\n";
}

void vWorkspace::addVectors(bool cpu, bool gpu, bool timeMeasure) const {
	int position1, position2, position3, position4;
	std::cout << "Which vector do you want to add: ";
	if(!inputPosition(&position1))
		return;

	std::cout << "With which vector: ";
	if(!inputPosition(&position2))
		return;
	if (cpu) {
		std::cout << "Chose position in workspace where result will be save: ";
		intInput(&position3);

		if (position3 >= _size || position3 < 0) {
			std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
			return;
		}
		if (_ptr[position3] != nullptr) {
			std::cout << "Given position is already taken!\n";
			return;
		}
	}

	if (gpu) {
		std::cout << "Chose position in workspace where result from GPU will be save: ";
		intInput(&position4);

		if (position4 >= _size || position4 < 0) {
			std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
			return;
		}
		if (_ptr[position4] != nullptr) {
			std::cout << "Given position is already taken!\n";
			return;
		}
	}
	
	if (cpu) {
		if (timeMeasure) {

		}
		_ptr[position3] = new Vector(_ptr[position2]->getSize());
		*_ptr[position3] = *_ptr[position1] + *_ptr[position2];
		if (timeMeasure) {

		}
	}
	
	if (gpu) {
		if (timeMeasure) {

		}

		if (timeMeasure) {

		}
	}
}

void vWorkspace::substractVectors(bool cpu, bool gpu, bool timeMeasure) const {
	int position1, position2, position3, position4;
	std::cout << "Which vector do you want to aubstract: ";
	if (!inputPosition(&position1))
		return;

	std::cout << "With which vector: ";
	if (!inputPosition(&position2))
		return;

	if(cpu){
	std::cout << "Chose position in workspace where result will be save: ";
	intInput(&position3);
		if (position3 >= _size || position3 < 0) {
			std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
			return;
		}
		if (_ptr[position3] != nullptr) {
			std::cout << "Given position is already taken!\n";
			return;
		}
	}

	if (gpu) {
		std::cout << "Chose position in workspace where result from GPU will be save: ";
		intInput(&position4);

		if (position4 >= _size || position4 < 0) {
			std::cout << "Invalid position choose number in range: 0 - " << _size - 1 << '\n';
			return;
		}
		if (_ptr[position4] != nullptr) {
			std::cout << "Given position is already taken!\n";
			return;
		}
	}
	
	if (cpu) {
		if (timeMeasure) {

		}
		_ptr[position3] = new Vector(_ptr[position2]->getSize());
		*_ptr[position3] = *_ptr[position1] - *_ptr[position2];
		if (timeMeasure) {

		}
	}
	
	if (gpu) {
		if (timeMeasure) {

		}

		if (timeMeasure) {

		}

	}
}

void vWorkspace::dotProductVectors(bool cpu, bool gpu, bool timeMeasure) const {
	int position1, position2;
	std::cout << "Which vector do you want to dot product: ";
	if (!inputPosition(&position1))
		return;

	std::cout << "With which vector: ";
	if (!inputPosition(&position2))
		return;

	if (cpu) {
		if (timeMeasure) {

		}
		std::cout << "Dot product = "<< *_ptr[position1] * *_ptr[position2];
		if (timeMeasure) {

		}
	}

	if (gpu) {
		if (timeMeasure) {

		}
		//std::cout << "Dot product = " << *_ptr[position1] * *_ptr[position2];
		if (timeMeasure) {

		}
	}
	
}

vWorkspace::vWorkspace(int size):
	_size(size),
	_ptr(new Vector*[size])
	{
	for (int i = 0; i < size; i++) {
		_ptr[i] = nullptr;
	}
}

vWorkspace::~vWorkspace() {
	for (int i = 0; i < _size; i++) {
		delete _ptr[i];
	}
	delete[] _ptr;
}

