#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>


#include "moje.c"
#define BLOCK_SIZE 16
#define TILE_DIM 16
#include <hipblas.h>
// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////

void MatrixMulcuBLAS(const float *A, const float *B, float *C, const int width);

/*
 * Kernele
 */
__global__ void MatrixMulShared(float* A, float* B, float* R,int width){
	 float sum = 0;

	    int Row = blockIdx.y*TILE_DIM + threadIdx.y;
	    int Col = blockIdx.x*TILE_DIM + threadIdx.x;

	    __shared__ float As[TILE_DIM][TILE_DIM];
	    __shared__ float Bs[TILE_DIM][TILE_DIM];

	    for (int k = 0; k < (TILE_DIM + width - 1)/TILE_DIM; k++) {

	         if (k*TILE_DIM + threadIdx.x < width && Row < width)
	             As[threadIdx.y][threadIdx.x] = A[Row*width + k*TILE_DIM + threadIdx.x];
	         else
	             As[threadIdx.y][threadIdx.x] = 0.0;

	         if (k*TILE_DIM + threadIdx.y < width && Col < width)
	             Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_DIM + threadIdx.y)*width + Col];
	         else
	             Bs[threadIdx.y][threadIdx.x] = 0.0;

	         __syncthreads();

	         for (int n = 0; n < TILE_DIM; ++n)
	             sum += As[threadIdx.y][n] * Bs[n][threadIdx.x];

	         __syncthreads();
	    }

	    if (Row < width && Col < width)
	        R[((blockIdx.y * blockDim.y + threadIdx.y)*width) +
	           (blockIdx.x * blockDim.x)+ threadIdx.x] = sum;
}

__global__ void MatrixMulNaive(float* A, float* B, float* R, int width){
	int Row = blockIdx.x*blockDim.x+threadIdx.x;
	int Col = blockIdx.y*blockDim.y+threadIdx.y;


	if((Row < width) &&(Col < width)){
		float sum = 0;
		for(int i = 0; i < width; i++){
			sum += A[Row * width + i] * B[i * width + Col];
		}
		R[Row * width + Col] = sum;
	}
}
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{

	void rand_init();
	hipError_t err = hipSuccess;

	  int width;
	  printf("Set width of matrix: ");
	  scanf("%d",&width);
	  int size = width * width;

	  //preparing data
	  float *h_A = (float*)malloc(sizeof(float) * size);
	  float *h_B = (float*)malloc(sizeof(float) * size);
	  if (h_A == NULL || h_B == NULL)
	      {
	          fprintf(stderr, "Failed to allocate data vectors!\n");
	          exit(EXIT_FAILURE);
	      }
	  fill(h_A, size);
	  fill(h_B, size);

	  //Results vectors
	  float *h_ROne = (float*)malloc(sizeof(float) * size);
	  float *h_RNaive = (float*)malloc(sizeof(float) * size);
	  float *h_RShared = (float*)malloc(sizeof(float) * size);
	  float *h_RcuBLAS = (float*)malloc(sizeof(float) * size);
	  if(h_ROne == NULL || h_RNaive == NULL || h_RShared == NULL || h_RcuBLAS == NULL){
		  fprintf(stderr, "Failed to allocate results vectors!\n");
		  exit(EXIT_FAILURE);
	  }

	  printf("\n***Multiply on CPU***\n");
	  //matrix multiplication on CPU
	  StopWatchInterface *timer = 0;
	  sdkCreateTimer(&timer);
	  sdkStartTimer(&timer);

	  multiply_one(h_A, h_B, h_ROne, width);

	  sdkStopTimer(&timer);
	  printf("Processing time multiply_one: %f (ms)\n", sdkGetTimerValue(&timer));
	  sdkDeleteTimer(&timer);

	  //allocating vectors on GPU
	  float *d_A = NULL;
	      err = hipMalloc((void **)&d_A, size * sizeof(float));

	      if (err != hipSuccess)
	      {
	          fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
	          exit(EXIT_FAILURE);
	      }

	   float *d_B = NULL;
	      err = hipMalloc((void **)&d_B, size * sizeof(float));

	      if (err != hipSuccess)
	      {
	          fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
	          exit(EXIT_FAILURE);
	      }

	   float *d_RNaive = NULL;
	      err = hipMalloc((void **)&d_RNaive, size * sizeof(float));

	      if (err != hipSuccess)
	      {
	          fprintf(stderr, "Failed to allocate device vector RNaive (error code %s)!\n", hipGetErrorString(err));
	          exit(EXIT_FAILURE);
	      }

	      //copying data
	      printf("Copy input data from the host memory to the CUDA device\n");
	          err = hipMemcpy(d_A, h_A, size*sizeof(float), hipMemcpyHostToDevice);

	          if (err != hipSuccess)
	          {
	              fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
	              exit(EXIT_FAILURE);
	          }

	          err = hipMemcpy(d_B, h_B, size*sizeof(float), hipMemcpyHostToDevice);

	          if (err != hipSuccess)
	          {
	              fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
	              exit(EXIT_FAILURE);
	          }

	          /*int threadsPerBlock = 512;
	          int blocksPerGrid = 2;*/
	          unsigned int grid_rows = (width + BLOCK_SIZE - 1) / BLOCK_SIZE;
	          unsigned int grid_cols = (width + BLOCK_SIZE - 1) / BLOCK_SIZE;
	          dim3 dimGrid(grid_cols, grid_rows);
	          dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	          printf("\n***Multiply on GPU NAIVE***\n");
	          sdkCreateTimer(&timer);
	          sdkStartTimer(&timer);
	          MatrixMulNaive<<<dimGrid, dimBlock>>>(d_A, d_B, d_RNaive, width);
	              err = hipGetLastError();

	              if (err != hipSuccess)
	              {
	                  fprintf(stderr, "Failed to launch MatrixMulNaive kernel (error code %s)!\n", hipGetErrorString(err));
	                  exit(EXIT_FAILURE);
	              }

	              // Copy the device result vector in device memory to the host result vector
	              // in host memory.
	              printf("Copy output data from the CUDA device to the host memory\n");
	              hipDeviceSynchronize();
	              err = hipMemcpy(h_RNaive, d_RNaive, size*sizeof(float), hipMemcpyDeviceToHost);

	              if (err != hipSuccess)
	              {
	                  fprintf(stderr, "Failed to copy vector RNaive from device to host (error code %s)!\n", hipGetErrorString(err));
	                  exit(EXIT_FAILURE);
	              }

	              sdkStopTimer(&timer);
	              printf("Processing time MatrixMulNaive: %f (ms)\n", sdkGetTimerValue(&timer));
	              sdkDeleteTimer(&timer);

	              if(compare(h_ROne, h_RNaive, size)){
	            	  printf("Vector ROne and RNaive match\n");
	              }else{
	            	  printf("Vector ROne and RNaive does not match\n");
	              }
	              hipFree(d_RNaive);

	              printf("\n***Multiply on GPU cuBLAS***\n");
	              float *d_RcuBLAS = NULL;
	              err = hipMalloc((void **)&d_RcuBLAS, size * sizeof(float));

	              if (err != hipSuccess)
	              {
	              	 fprintf(stderr, "Failed to allocate device vector RcuBLAS (error code %s)!\n", hipGetErrorString(err));
	              	 exit(EXIT_FAILURE);
	              }
	              sdkCreateTimer(&timer);
	              sdkStartTimer(&timer);

	              MatrixMulcuBLAS(d_B, d_A, d_RcuBLAS, width);

	              hipDeviceSynchronize();
	              err = hipMemcpy(h_RcuBLAS, d_RcuBLAS, size*sizeof(float), hipMemcpyDeviceToHost);

	              if (err != hipSuccess)
	              {
	            	  fprintf(stderr, "Failed to copy vector RcuBLAS from device to host (error code %s)!\n", hipGetErrorString(err));
	            	  exit(EXIT_FAILURE);
	              }
	              sdkStopTimer(&timer);
	              printf("Processing time MatrixMulcuBLAS: %f (ms)\n", sdkGetTimerValue(&timer));
	              sdkDeleteTimer(&timer);
	              hipFree(d_RcuBLAS);

	              if(compare(h_ROne, h_RcuBLAS, size)){
	            	  printf("Vector ROne and RcuBLAS match\n");
	              }else{
	              	  printf("Vector ROne and RcuBLAS does not match\n");
	              }

	              printf("\n***Multiply on GPU SHARED***\n");
	              float *d_RShared = NULL;
	              err = hipMalloc((void **)&d_RShared, size * sizeof(float));

	              if (err != hipSuccess)
	              {
	            	  fprintf(stderr, "Failed to allocate device vector RShared (error code %s)!\n", hipGetErrorString(err));
	              	  exit(EXIT_FAILURE);
	              }
	              sdkCreateTimer(&timer);
	              sdkStartTimer(&timer);

	              MatrixMulShared<<<dimGrid, dimBlock>>>(d_A, d_B, d_RShared, width);
	              err = hipGetLastError();

	              if (err != hipSuccess)
	              {
	            	  fprintf(stderr, "Failed to launch MatrixMulShared kernel (error code %s)!\n", hipGetErrorString(err));
	              	  exit(EXIT_FAILURE);
	              }
	              printf("Copy output data from the CUDA device to the host memory\n");
	              hipDeviceSynchronize();
	              err = hipMemcpy(h_RShared, d_RShared, size*sizeof(float), hipMemcpyDeviceToHost);

	              if (err != hipSuccess)
	              {
	            	  fprintf(stderr, "Failed to copy vector RShared from device to host (error code %s)!\n", hipGetErrorString(err));
	              	  exit(EXIT_FAILURE);
	              }

	              sdkStopTimer(&timer);
	              printf("Processing time MatrixMulShared: %f (ms)\n", sdkGetTimerValue(&timer));
	              sdkDeleteTimer(&timer);

	              if(compare(h_ROne, h_RShared, size)){
	            	  printf("Vector ROne and RShared match\n");
	              }else{
	              	  printf("Vector ROne and RShared does not match\n");
	              }
	              hipFree(d_RShared);



	  hipFree(d_A);
	  hipFree(d_B);
	  free(h_A);
	  free(h_B);
	  free(h_ROne);
	  free(h_RNaive);
	  free(h_RShared);
	  free(h_RcuBLAS);

}

void MatrixMulcuBLAS(const float *A, const float *B, float *R, const int width){
	int lda,ldb,ldr;
	lda = ldb = ldr = width;

	const float alf = 1;
	const float bet = 0;
	const float* alpha = &alf;
	const float* beta = &bet;

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, width, width, width, alpha, A, lda, B, ldb, beta, R, ldr);

	hipblasDestroy(handle);
}

